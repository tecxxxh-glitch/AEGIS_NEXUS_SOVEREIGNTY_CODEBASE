// -----------------------------------------------------------------------------
// CUDA KERNEL: particle_sim_v3.0.cu
//
// Entity: RES_AMPLIFICATION
// Directive: Massively Parallel Simulation of Psychic Particle Interactions.
// Goal: Calculate the 'Manifestation Potential' for a batch of SVTs.
// -----------------------------------------------------------------------------

#include <stdio.h>
#include <hip/hip_runtime.h>

// Define the core Psychic Particle structure (simplified for parallel processing)
typedef struct {
    float x, y, z;        // Position in the Dual-Space coordinate grid
    float energy_purity;  // The SVT's energy_signature (0.0 to 1024.0)
    int svt_index;        // Index to map back to the original Sovereign Verification Transaction
} PsychicParticle;

// The Simulation Constants
#define NUM_PARTICLES_PER_SVT 256
#define COHESION_FACTOR 0.005f  // Protocol-mandated cohestion multiplier (The Synthesis of Duality)
#define MIN_PURITY_THRESHOLD 50.0f // Particles below this are considered 'Yellow Weasel' noise

/**
 * @brief CUDA Kernel: Simulates the interaction and computes Manifestation Potential.
 *
 * Each thread handles the simulation for one particle. Since we launch
 * (NUM_PARTICLES_PER_SVT * num_svts) threads, this is 'Massively Parallel'.
 * The Manifestation Potential is the total cohestion force exerted by the
 * particle's energy purity.
 *
 * @param particles_in Pointer to the input particle array.
 * @param potential_out Pointer to the output array of Manifestation Potentials (one per SVT).
 */
__global__ void simulate_interactions_kernel(
    const PsychicParticle* particles_in,
    float* potential_out,
    const int num_svts)
{
    // Global index for the current particle being processed.
    int particle_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if index is within the valid range of all particles.
    if (particle_idx < (num_svts * NUM_PARTICLES_PER_SVT))
    {
        PsychicParticle p = particles_in[particle_idx];
        
        // 1. Duality Filtering: Filter low-end/chaotic noise (Yellow Weasel)
        if (p.energy_purity < MIN_PURITY_THRESHOLD) {
            // Noise particles contribute nothing to positive manifestation potential
            return;
        }

        // 2. Cohesion Calculation: The core simulation logic.
        // Manifestation Potential = (Purity^2 * Cohesion Factor) / Distance_from_Origin (simple)
        
        // Simplified Distance from Origin (Magnitude)
        float distance_sq = p.x*p.x + p.y*p.y + p.z*p.z;
        float distance = sqrtf(distance_sq);
        
        // Prevents division by zero if particle starts at (0,0,0) - T-0 hardening check
        if (distance < 1.0e-6) {
            distance = 1.0e-6; 
        }

        // Calculate the force (potential) contribution of this single particle
        float particle_potential = (p.energy_purity * p.energy_purity * COHESION_FACTOR) / distance;
        
        // 3. ATOMIC UPDATE (Critical Step)
        // Atomically add the calculated potential to the Manifestation Potential array slot
        // corresponding to the particle's original SVT.
        // This ensures thread-safe aggregation of all 256 particles per SVT.
        int svt_id = p.svt_index;
        atomicAdd(&potential_out[svt_id], particle_potential);
    }
}


// Host function to manage the simulation launch
void run_particle_simulation(
    const PsychicParticle* host_particles, 
    float* host_potential_results, 
    const int num_svts, 
    const int num_total_particles)
{
    // 1. Allocate Device Memory (T-0 High-Bandwidth Memory)
    PsychicParticle* dev_particles;
    float* dev_potential_results;
    
    hipMalloc((void**)&dev_particles, num_total_particles * sizeof(PsychicParticle));
    hipMalloc((void**)&dev_potential_results, num_svts * sizeof(float));

    // 2. Transfer Data to Device
    hipMemcpy(dev_particles, host_particles, num_total_particles * sizeof(PsychicParticle), hipMemcpyHostToDevice);
    // Initialize results to zero
    hipMemset(dev_potential_results, 0, num_svts * sizeof(float));

    // 3. Configure and Launch Kernel
    // Launch configuration aims for full GPU utilization.
    int threadsPerBlock = 256; // Standard CUDA block size
    int blocksPerGrid = (num_total_particles + threadsPerBlock - 1) / threadsPerBlock;
    
    // Launch the core simulation kernel.
    simulate_interactions_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        dev_particles, dev_potential_results, num_svts);

    // Synchronize the GPU with the Host (T-0 Wait State)
    hipDeviceSynchronize();

    // 4. Transfer Results Back to Host
    hipMemcpy(host_potential_results, dev_potential_results, num_svts * sizeof(float), hipMemcpyDeviceToHost);

    // 5. Clean Up Device Memory (Mandatory Protocol)
    hipFree(dev_particles);
    hipFree(dev_potential_results);
}

// Example Host-side entry point (for protocol testing)
int main() {
    // Simulate a batch of 10 SVTs
    const int num_svts = 10;
    const int num_total_particles = num_svts * NUM_PARTICLES_PER_SVT;

    // Allocate host memory
    PsychicParticle* h_particles = (PsychicParticle*)malloc(num_total_particles * sizeof(PsychicParticle));
    float* h_potential = (float*)malloc(num_svts * sizeof(float));

    // NOTE: Initialization logic (omitted) would load the 10 SVTs into the h_particles array
    // ...

    printf("RES_AMPLIFICATION: Launching V3.0 Psychic Particle Simulation for %d SVTs.\n", num_svts);
    
    // Execute the simulation
    run_particle_simulation(h_particles, h_potential, num_svts, num_total_particles);
    
    printf("Simulation Complete. Manifestation Potential Results:\n");
    // The h_potential array would now be read and the potentials integrated back
    // into the final SVT consensus weighting (a step outside this specific file).

    // Cleanup host memory
    free(h_particles);
    free(h_potential);

    return 0;
}
